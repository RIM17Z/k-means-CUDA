#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include ""
#include "hip/hip_vector_types.h"
#include <thrust/device_vector.h>
#include <float.h>
#include <stdio.h>
#include "KMeansTypes.h"

struct sum_float4 : public thrust::binary_function<float4, float4, float4> {
	__host__ __device__ float4 operator()(float4 x, float4 y) { return make_float4(x.x + y.x, x.y + y.y, x.z + y.z, 0.0); }
};
struct equal_id : public thrust::binary_function<float4, float4, bool>{
	__host__ __device__ bool operator()(const float4 a, const float4 b) const { return *((char*)&(a.w) + 3) == *((char*)&(b.w) + 3); }
};

struct get_keys : public thrust::unary_function<float4, int>{
	__host__ __device__ int operator()(const float4 x) { return *((char*)&(x.w) + 3); }
};

__device__ bool d_converged[1];

__global__ void assignKernel(float4* d_vertices, float4* d_centroids, int V, int C)
{
	float distold = FLT_MAX;
	__shared__ float4 s_centroids[MAX_CLUSTERS_CNT];
	float4 p;
	unsigned int a = 0;
	unsigned int j;

	unsigned int idx = (blockIdx.x * blockDim.x + threadIdx.x);

	if (idx == 0)
		d_converged[0] = true;
	
	s_centroids[threadIdx.x] = d_centroids[threadIdx.x];

	__syncthreads();

	if (idx < V){
		p = d_vertices[idx];

		for (j = 0; j < C; j++){
			float tmp_x = s_centroids[j].x - p.x;
			float tmp_y = s_centroids[j].y - p.y;
			float tmp_z = s_centroids[j].z - p.z;
			float distnew = (tmp_x*tmp_x + tmp_y*tmp_y + tmp_z*tmp_z);
			if (distold > distnew){
				a = j;
				distold = distnew;
			}
		}
		//__syncthreads();
		if (p.w != s_centroids[a].w)
			d_converged[0] = false;
		d_vertices[idx].w = s_centroids[a].w;
	}

}

__global__ void sumClustersKernel(float4* d_vertices, float3* d_sums, int* d_clusters_cnt, int V, int C){

	unsigned int idx = (blockIdx.x * blockDim.x + threadIdx.x);
	if (idx < V){
		float4 vertex = d_vertices[idx];
		unsigned int id = *((char*)&(vertex.w) + 3);
		__syncthreads();
		atomicAdd(&(d_sums[id].x), vertex.x);
		atomicAdd(&(d_sums[id].y), vertex.y);
		atomicAdd(&(d_sums[id].z), vertex.z);
		atomicAdd(&d_clusters_cnt[id], 1);
	}
}

__global__ void moveCentroidsKernel(float4* d_centroids, float3* d_sums, int* d_clusters_cnt, int C)
{
	unsigned int idx = (blockIdx.x*blockDim.x + threadIdx.x);
	
	if (idx < C){
		int cnt = d_clusters_cnt[idx];
		if (cnt > 0){
			d_centroids[idx].x = d_sums[idx].x / cnt;
			d_centroids[idx].y = d_sums[idx].y / cnt;
			d_centroids[idx].z = d_sums[idx].z / cnt;

		}
	}
}
//512,512
extern "C" bool assignPoints(KMeans::DataPoint* d_vertices, KMeans::DataPoint* d_centroids, int V, int C, int grid_size, int block_size)
{
	bool converged[1];
	bool *ptr_d_converged;
	assignKernel << < (V + grid_size - 1) / grid_size, block_size >> >((float4*)d_vertices, (float4*)d_centroids, V, C);
	hipGetSymbolAddress((void**)&ptr_d_converged, d_converged);
	checkCudaErrors(hipMemcpy(converged, ptr_d_converged, sizeof(bool), hipMemcpyDeviceToHost));
	return converged[0];
}
//512,512
extern "C" void sumClusters(KMeans::DataPoint* d_vertices, KMeans::Pos* d_sums, int* d_clusters_cnt, int V, int C, int grid_size, int block_size){
	/*
	thrust::device_ptr<float4> d_v_ptr = thrust::device_pointer_cast((float4*)d_vertices);
	thrust::device_ptr<float3> d_sums_ptr = thrust::device_pointer_cast((float4*)d_sums);
	thrust::device_ptr<int> d_sum_id_ptr = thrust::device_pointer_cast(d_sum_id);
	thrust::device_ptr<int> d_keys_ptr = thrust::device_pointer_cast(d_keys);
	thrust::equal_to<int> binary_pred;
	thrust::transform(d_v_ptr, d_v_ptr + V, d_keys_ptr, get_keys());
	thrust::reduce_by_key(d_keys_ptr, d_keys_ptr + V, d_v_ptr, d_sum_id_ptr, d_sums_ptr, binary_pred, sum_float4());
	*/
	sumClustersKernel << < (V + grid_size - 1) / grid_size, block_size >> >((float4*)d_vertices, (float3*)d_sums, d_clusters_cnt, V, C);
}
//256,256
extern "C" void moveCentroids(KMeans::DataPoint* d_centroids, KMeans::Pos* d_sums, int* d_clusters_cnt, int C, int grid_size, int block_size)
{
	moveCentroidsKernel << < (C + grid_size - 1) / grid_size, block_size >> >((float4*)d_centroids, (float3*)d_sums, d_clusters_cnt, C);
}
#endif // #ifndef _KERNEL_H_
