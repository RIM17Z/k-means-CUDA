#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_vector_types.h"
#include <thrust/device_vector.h>
#include <float.h>
#include <stdio.h>
#include "KMeansTypes.h"

__device__ bool d_converged;

struct sum_float4 : public thrust::binary_function<float4, float4, float4> {
	__host__ __device__ float4 operator()(float4 x, float4 y) { return make_float4(x.x + y.x, x.y + y.y, x.z + y.z, 0.0); }
};
struct equal_id : public thrust::binary_function<float4, float4, bool>{
	__host__ __device__ bool operator()(const float4 a, const float4 b) const { return *((char*)&(a.w) + 3) == *((char*)&(b.w) + 3); }
};

struct get_keys : public thrust::unary_function<float4, int>{
	__host__ __device__ int operator()(const float4 x) { return *((char*)&(x.w) + 3); }
};

__global__ void assignKernel(float4* d_vertices, float4* d_centroids, int V, int C)
{
	float distold = FLT_MAX;
	__shared__ float4 s_centroids[512];
	float4 p;
	unsigned int a = 0;
	unsigned int j;

	unsigned int idx = (blockIdx.x * blockDim.x + threadIdx.x);

	if (idx == 0)
		d_converged = true;
	
	//TODO allocate d_centroids of size blockDim.x and fill excess with zeros
	// then remove if clause
	if (threadIdx.x < C)
		s_centroids[threadIdx.x] = d_centroids[threadIdx.x];
	else
		s_centroids[threadIdx.x] = make_float4(0.0, 0.0, 0.0, 0.0);

	// TODO same as for centroids
	if (idx < V)
		p = d_vertices[idx];
	else
		p = make_float4(0.0, 0.0, 0.0, 0.0);

	__syncthreads();

	if (idx < V){
		for (j = 0; j < C; j++){
			float tmp_x = s_centroids[j].x - p.x;
			float tmp_y = s_centroids[j].y - p.y;
			float tmp_z = s_centroids[j].z - p.z;
			float distnew = (tmp_x*tmp_x + tmp_y*tmp_y + tmp_z*tmp_z);
			if (distold > distnew){
				a = j;
				distold = distnew;
			}
		}
		__syncthreads();
		if (*((char*)&(p.w) + 3) != *((char*)&(s_centroids[j].w) + 3))
			d_converged = false;
		d_vertices[idx].w = s_centroids[a].w;
	}

}

__global__ void sumClustersKernel(float4* d_vertices, float3* d_sums, int* d_clusters_cnt, int V, int C){

	unsigned int idx = (blockIdx.x * blockDim.x + threadIdx.x);
	if (idx < V){
		float4 vertex = d_vertices[idx];
		unsigned int id = *((char*)&(vertex.w) + 3);
		__syncthreads();
		atomicAdd(&(d_sums[id].x), vertex.x);
		atomicAdd(&(d_sums[id].y), vertex.y);
		atomicAdd(&(d_sums[id].z), vertex.z);
		atomicAdd(&d_clusters_cnt[id], 1);
	}
}

__global__ void moveCentroidsKernel(float4* d_centroids, float3* d_sums, int* d_clusters_cnt, int C)
{
	unsigned int idx = (blockIdx.x*blockDim.x + threadIdx.x);
	
	if (idx < C){
		int cnt = d_clusters_cnt[idx];
		if (cnt > 0){
			d_centroids[idx].x = d_sums[idx].x / cnt;
			d_centroids[idx].y = d_sums[idx].y / cnt;
			d_centroids[idx].z = d_sums[idx].z / cnt;

		}
	}
}

extern "C" bool assignPoints(KMeans::DataPoint* d_vertices, KMeans::DataPoint* d_centroids, int V, int C)
{
	bool converged = false;
	assignKernel <<< (V + 511) / 512, 512 >>>((float4*)d_vertices, (float4*)d_centroids, V, C);
	//hipMemcpyFromSymbol(&converged, HIP_SYMBOL((void*)) d_converged, sizeof(converged), 0, hipMemcpyDeviceToHost);
	return converged;
}

extern "C" void sumClusters(KMeans::DataPoint* d_vertices, KMeans::Pos* d_sums, int* d_clusters_cnt, int V, int C){

	//thrust::device_ptr<float4> d_v_ptr = thrust::device_pointer_cast((float4*)d_vertices);
	//thrust::device_ptr<float4> d_sums_ptr = thrust::device_pointer_cast(d_sums);
	//thrust::device_ptr<int> d_sum_id_ptr = thrust::device_pointer_cast(d_sum_id);
	//thrust::device_ptr<int> d_keys_ptr = thrust::device_pointer_cast(d_keys);
	//thrust::equal_to<int> binary_pred;
	//thrust::transform(d_v_ptr, d_v_ptr + V, d_keys_ptr, get_keys());
	//thrust::reduce_by_key(d_keys_ptr, d_keys_ptr + V, d_v_ptr, d_sum_id_ptr, d_sums_ptr, binary_pred, sum_float4());
	sumClustersKernel << < (V + 511) / 512, 512 >> >((float4*)d_vertices, (float3*)d_sums, d_clusters_cnt, V, C);
}

extern "C" void moveCentroids(KMeans::DataPoint* d_centroids, KMeans::Pos* d_sums, int* d_clusters_cnt, int C)
{
	moveCentroidsKernel << < (C + 255) / 256, 256 >> >((float4*)d_centroids, (float3*)d_sums, d_clusters_cnt, C);
}
#endif // #ifndef _KERNEL_H_
